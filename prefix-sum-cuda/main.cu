#include "hip/hip_runtime.h"
/********************************************//** 
 * Calculate exclusive scan, find repeats, 
 * Output remainings in parallel with GPU
 * Using CUDA language
 *
 * Written by:
 * Dongyang Yao (dongyang.yao@rutgers.edu)
 ***********************************************/

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <chrono>
#include <string>
#include <fstream>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

using namespace std::chrono;

int N = 1000000;

#define THREADS_PER_BLK 128

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS)) 

/* Get random inputs */
void generate_randoms(int* randoms, int length, int max) {
  std::srand(std::time(0));
  for (int i = 0; i < length; i++) {
    randoms[i] = std::rand() % max;
    //std::cout << randoms[i] << std::endl;
  }
}

/* Print out the numbers */
void show_samples(int* numbers, int count) {
  for (int i = 0; i < count; i++) std::cout << numbers[i] << std::endl;
}

/* Get exclusive scan in sequential */
void generate_exclusive_scan_truth(int* check, int* randoms, int length) {
  check[0] = 0;
  for (int i = 1; i < length; i++)
    check[i] = check[i - 1] + randoms[i - 1];
}

/* Get repeats in sequential */
int generate_find_repeats_truth(std::vector<int>* check, int* randoms, int length) {
  int count = 0;
  for (int i = 0; i < length - 1; i++) {
    if (randoms[i] == randoms[i + 1]) {
      count++;
      check->push_back(i);
    }
  }
  return count;
}

/* Get remainings in sequential */
int generate_remove_repeats_truth(std::vector<int>* check, int* randoms, int length) {
  int count = 0;
  for (int i = 0; i < length - 1; i++) {
    if (randoms[i] != randoms[i + 1]) {
      count++;
      check->push_back(randoms[i]);
    }
  }
  count++;
  check->push_back(randoms[length - 1]);
  return count;
}

/* Compare result with truth */
void check_results(int* results, int* check, int length) {
  bool result = true;
  for (int i = 0; i < length; i++) {
    //std::cout << check[i] << " " << results[i] << std::endl;
    if (!(check[i] == results[i])) {
      std::cout << "mis-match at " << i << std::endl;
      result = false;
    }
  }

  if (result) std::cout << "pass successfully" << std::endl;
  else std::cout << "you have error shown above" << std::endl;
}

/* Compare result with truth */
void check_results(int* results, std::vector<int>* check) {
  bool result = true;

  int i = 0;
  for (std::vector<int>::iterator iter = check->begin(); iter != check->end(); iter++) {
    if (!(*iter == results[i])) {
      std::cout << *iter << " " << results[i] << std::endl;
      std::cout << "mis-match at " << i << std::endl;
      result = false;
    }
    i++;
  }

  if (result) std::cout << "pass successfully" << std::endl;
  else std::cout << "you have error shown above" << std::endl;
}

/* Get CUDA info on this computer */
void get_cuda_info() {
  int device_count = 0;
  hipError_t error = hipGetDeviceCount(&device_count);

  std::cout << "number of gpu: " << device_count << std::endl;

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, i);
    std::cout << "name: " << device_props.name << std::endl;
  }
}

/* Get next POW of 2 */
int get_next_pow_2(int n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  n++;
  return n;
}

/* Get exclusive scan on GPU */
__global__ void exclusive_scan_gpu(int* input, int* output, int n) {
  __shared__ int temp[4 * THREADS_PER_BLK];
  int thid_global = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  int thid = threadIdx.x;

  {  
    int offset = 1;
    //temp[2 * thid] = input[2 * thid_global];
    //temp[2 * thid + 1] = input[2 * thid_global + 1];
    
    int aind = thid;
    int bind = thid + n / 2;
    int bankOffsetA = CONFLICT_FREE_OFFSET(aind);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bind);
    temp[aind + bankOffsetA] = input[thid_global];
    temp[bind + bankOffsetB] = input[thid_global + n / 2];  
     

    for (int d = n >> 1; d > 0; d >>= 1) {
      __syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        ai += CONFLICT_FREE_OFFSET(ai);
        bi += CONFLICT_FREE_OFFSET(bi);
        temp[bi] += temp[ai];
      }
      offset *= 2;
    }

    if (thid == 0) { 
      //temp[n - 1] = 0;
      temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d *= 2) {
      offset >>= 1;
      __syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        ai += CONFLICT_FREE_OFFSET(ai);
        bi += CONFLICT_FREE_OFFSET(bi);
        int t = temp[ai];
        temp[ai] = temp[bi];
        temp[bi] += t;
      }
    }

    __syncthreads();
    //output[2 * thid_global] = temp[2 * thid];
    //output[2 * thid_global + 1] = temp[2 * thid + 1];
    //printf("%d:%d %d:%d\n", 2 * thid_global, output[2 * thid_global], 2 * thid_global + 1, output[2 * thid_global + 1]);
    output[thid_global] = temp[aind + bankOffsetA];
    output[thid_global + n / 2] = temp[bind + bankOffsetB];
  }

}

/* Add partial results with base to get full result on GPU */
__global__ void add_base_gpu(int* device_input, int* device_output, int block_index) {
  int block_last_element = block_index * THREADS_PER_BLK * 2 - 1;
  
  int base = device_input[block_last_element] + device_output[block_last_element];
  
  int thid = block_index * blockDim.x + threadIdx.x;

  device_output[2 * thid] += base;
  device_output[2 * thid + 1] += base;
}

/* Mark repeat on GPU */
__global__ void mark_flags_gpu(int* input, int* flags, int length, bool mark_repeat) {
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if (thid < length - 1) {
     if (input[thid] == input[thid + 1]) {
       flags[thid] = mark_repeat ? 1 : 0;
     } else {
       flags[thid] = mark_repeat ? 0 : 1;
     }
     //printf("id:%d %d\n", thid, flags[thid]);
  }
}

/* Get repeats on GPU */
__global__ void get_repeat_results(int* input, int* flags_scaned, int length, int* output, bool mark_repeat) {
  int thid = blockIdx.x * blockDim.x + threadIdx.x;
  if ((thid < length - 1) && (flags_scaned[thid] < flags_scaned[thid + 1])) {
    //printf("id:%d %d\n", thid, flags_scaned[thid]);
    output[flags_scaned[thid]] = mark_repeat ? thid : input[thid]; 
  }
  if ((thid == length - 1) && (!mark_repeat)) {
    //printf("id:%d %d\n", thid, flags_scaned[thid]);
    output[flags_scaned[thid]] = input[length - 1];
    
  }
}

/* Get exclusive scan on CPU */
void exclusive_scan_sequential(int* randoms, int length, int* output) {
    memmove(output, randoms, length * sizeof(int));

    // Upsweep phase
    for (int twod = 1; twod < length; twod*=2)
    {
        int twod1 = twod*2;
        // Parallel
        for (int i = 0; i < length; i += twod1)
        {
            output[i+twod1-1] += output[i+twod-1];
        }
    }

    output[length-1] = 0;

    // Downsweep phase
    for (int twod = length/2; twod >= 1; twod /= 2)
    {
        int twod1 = twod*2;
        // Parallel
        for (int i = 0; i < length; i += twod1)
        {
            int tmp = output[i+twod-1];
            output[i+twod-1] = output[i+twod1-1];
            output[i+twod1-1] += tmp;
        }
    }
}

/* Get repeats on CPU */
int find_repeats_sequential(int* results, int* randoms, int length) {
  int count = 0;

  int* flags = new int[length];

  for (int i = 0; i < length - 1; i++) {
    if (randoms[i] == randoms[i + 1]) {
      count++;
      flags[i] = 1;
    } else {
      flags[i] = 0;
    }
  }

  //for (int i = 0; i < length; i++) std::cout << flags[i] << std::endl;
  
  int length_rounded = get_next_pow_2(N);
  int* flags_scaned = new int[length_rounded];  

  exclusive_scan_sequential(flags, length_rounded, flags_scaned);

  //for (int i = 0; i < length; i++) std::cout << flags_scaned[i] << std::endl;

  for (int i = 0; i < length - 1; i++) {
    if (flags_scaned[i] < flags_scaned[i + 1]) {
      results[flags_scaned[i]] = i;
    }
  }

  delete[] flags;
  delete[] flags_scaned;

  return count;
}

int main(int argc, char** argv) {

  int* randoms;
  bool use_external = false;
  
  if (argc == 2) {
     int in = atoi(argv[1]);
     if (in != 0) N = in;
     else {
       std::string line;
       std::ifstream file (argv[1]);
       if (file.is_open()) {
         use_external = true;

         std::cout << "loading external data..." << std::endl;
	 getline(file, line);
	 N = std::stoi(line);
	 randoms = new int[N];

	 int i = 0;
	 while (getline(file, line)) {
	   randoms[i++] = std::stoi(line);   
	 }
	 
	 file.close();
       } else {
         std::cout << "cannot find the file!" << std::endl;
       }      
     }
  }

  std::cout << "**********" << std::endl;
  std::cout << "DEBUG INFO" << std::endl;
  std::cout << "**********" << std::endl;    

  std::cout << "number of threads per block: " << THREADS_PER_BLK << std::endl;
  
  const int MAX = 100;
  //const int NUM_SAMPLE = 10;

  if (!use_external) {  
    randoms = new int[N];
  
    std::cout << "generating random numbers..." << std::endl;
    std::cout << "max: " << MAX << std::endl;
  
    generate_randoms(randoms, N, MAX);
  }

  std::cout << "count: " << N << std::endl;
  
  //for (int i = 0; i < N; i++) std::cout << i << ":" << randoms[i] << std::endl;

  /*
  std::cout << "showing random numbers..." << std::endl;
  std::cout << "count: " << NUM_SAMPLE << std::endl;

  show_samples(randoms, NUM_SAMPLE);
  */

  int* exclusive_scan_check = new int[N];

  std::cout << "generating exclusive scan ground truth..." << std::endl;

  generate_exclusive_scan_truth(exclusive_scan_check, randoms, N);

  std::cout << "computing exclusive scan in cpu..." << std::endl;

  int length = get_next_pow_2(N);
  int* output_sequential = new int[length];
  
  high_resolution_clock::time_point start = high_resolution_clock::now();  
  
  exclusive_scan_sequential(randoms, length, output_sequential);
  
  high_resolution_clock::time_point end = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " <<  time_span.count() * 1000 << " ms" << std::endl;  
  
  check_results(output_sequential, exclusive_scan_check, N);

  /*
  std::cout << "showing ground truth..." << std::endl;
  std::cout << "count: " << NUM_SAMPLE << std::endl;
  
  show_samples(exclusive_scan_check, NUM_SAMPLE);
  */

  std::cout << "checking gpu availability..." << std::endl;
  get_cuda_info();
  
  std::cout << "rounding up to the next highest power of 2..." << std::endl;
  std:: cout << "rounded length: " << length << std::endl;
  
  int* exclusive_scan_gpu_results = new int[N];
  int* device_input;
  int* device_output;

  int* find_repeat_gpu_results = new int[N];
  int* flags;
  int* flags_scaned;
  int* find_repeat_output;

  int* remove_repeat_gpu_results = new int[N];
  int* flags_remain;
  int* flags_remain_scaned;
  int* remove_repeat_output;
  
  std::cout << "allocateing memory on gpu for input and output..." << std::endl;
  hipMalloc((void **) &device_input, sizeof(int) * length);
  hipMalloc((void **) &device_output, sizeof(int) * length);

  hipMalloc((void **) &flags, sizeof(int) * length);
  hipMalloc((void **) &flags_scaned, sizeof(int) * length);
  hipMalloc((void **) &find_repeat_output, sizeof(int) * length);	

  hipMalloc((void **) &flags_remain, sizeof(int) * length);
  hipMalloc((void **) &flags_remain_scaned, sizeof(int) * length);
  hipMalloc((void **) &remove_repeat_output, sizeof(int) * length);

  std::cout << "copying the random numbers from cpu to gpu..." << std::endl;
  hipMemcpy(device_input, randoms, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(device_output, randoms, sizeof(int) * N, hipMemcpyHostToDevice);

  std::cout << "computing exclusive scan on gpu..." << std::endl;
  
  int num_block = length / (THREADS_PER_BLK * 2);
  if (num_block == 0) num_block = 1;
  
  std::cout << "number of block: " << num_block << std::endl;

  start = high_resolution_clock::now();  

  exclusive_scan_gpu<<<num_block, THREADS_PER_BLK>>>(device_input, device_output, length / num_block);
  hipDeviceSynchronize();

  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  //hipMemcpy(exclusive_scan_gpu_results, device_output, sizeof(int) * N, hipMemcpyDeviceToHost);

  //for (int i = 0; i < N; i++) std::cout << i << ":" << exclusive_scan_gpu_results[i] << ":" << exclusive_scan_check[i] << std::endl;
  
  //std::cout << "multi-block" << std::endl;

  for (int i = 1; i < num_block; i++)
    add_base_gpu<<<1, THREADS_PER_BLK>>>(device_input, device_output, i);

  hipMemcpy(exclusive_scan_gpu_results, device_output, sizeof(int) * N, hipMemcpyDeviceToHost);
  
  //for (int i = 0; i < N; i++) std::cout << i << ":" << exclusive_scan_gpu_results[i] << ":" << exclusive_scan_check[i] << std::endl;

  check_results(exclusive_scan_gpu_results, exclusive_scan_check, N);

  std::cout << "computing exclusive scan using THRUST library..." << std::endl;

  int* scan_thrust_results = new int[N];

  thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
  thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
  
  hipMemcpy(d_input.get(), randoms, N * sizeof(int), hipMemcpyHostToDevice);

  start = high_resolution_clock::now();

  thrust::exclusive_scan(d_input, d_input + length, d_output);

  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  hipMemcpy(scan_thrust_results, d_output.get(), N * sizeof(int), hipMemcpyDeviceToHost);

  check_results(scan_thrust_results, exclusive_scan_check, N);

  std::cout << "generating find repeats ground truth..." << std::endl;
  
  std::vector<int>* find_repeats_check = new std::vector<int>(); 
  
  int repeats_count = generate_find_repeats_truth(find_repeats_check, randoms, N);

  std::cout << "computing find repeats on cpu..." << std::endl;

  int* find_repeats_sequential_results = new int[N];

  start = high_resolution_clock::now();

  find_repeats_sequential(find_repeats_sequential_results, randoms, N);

  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  check_results(find_repeats_sequential_results, find_repeats_check);

  std::cout << "number of repeats: " << repeats_count << std::endl;
  
  std::cout << "computing find repeats on gpu..." << std::endl;

  int num_block_repeat = length / THREADS_PER_BLK;
  if (num_block_repeat == 0) num_block_repeat = 1;

  start = high_resolution_clock::now();

  mark_flags_gpu<<<num_block_repeat, THREADS_PER_BLK>>>(device_input, flags, length, true);

  exclusive_scan_gpu<<<num_block, THREADS_PER_BLK>>>(flags, flags_scaned, length / num_block);
  hipDeviceSynchronize();
  
  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  for (int i = 1; i < num_block; i++)
    add_base_gpu<<<1, THREADS_PER_BLK>>>(flags, flags_scaned, i);

  get_repeat_results<<<num_block_repeat, THREADS_PER_BLK>>>(device_input, flags_scaned, N, find_repeat_output, true);
  hipMemcpy(find_repeat_gpu_results, find_repeat_output, sizeof(int) * N, hipMemcpyDeviceToHost);

  check_results(find_repeat_gpu_results, find_repeats_check);

  std::cout << "generating remove repeats ground truth..." << std::endl;
  
  std::vector<int>* remove_repeats_check = new std::vector<int>();

  start = high_resolution_clock::now();

  int remain_count = generate_remove_repeats_truth(remove_repeats_check, randoms, N);

  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  std::cout << "number of remains: " << remain_count << std::endl;

  std::cout << "computing remove repeats on gpu..." << std::endl;

  start = high_resolution_clock::now();

  mark_flags_gpu<<<num_block_repeat, THREADS_PER_BLK>>>(device_input, flags_remain, length, false);

  exclusive_scan_gpu<<<num_block, THREADS_PER_BLK>>>(flags_remain, flags_remain_scaned, length / num_block);
  hipDeviceSynchronize();

  end = high_resolution_clock::now();
  time_span = duration_cast<duration<double> >(end - start);
  std::cout << "elapsed time: " << time_span.count() * 1000 << " ms" << std::endl;

  for (int i = 1; i < num_block; i++)
    add_base_gpu<<<1, THREADS_PER_BLK>>>(flags_remain, flags_remain_scaned, i);

  get_repeat_results<<<num_block_repeat, THREADS_PER_BLK>>>(device_input, flags_remain_scaned, N, remove_repeat_output, false);

  hipMemcpy(remove_repeat_gpu_results, remove_repeat_output, sizeof(int) * N, hipMemcpyDeviceToHost);

  check_results(remove_repeat_gpu_results, remove_repeats_check);

  std::cout << "************" << std::endl;
  std::cout << "REQUIREMENTS" << std::endl;
  std::cout << "************" << std::endl;

  std::cout << "array A (exclusive scan)" << std::endl;
  std::cout << "size: " << N << std::endl;
  std::cout << "last element: " << exclusive_scan_gpu_results[N - 1] << std::endl;
  
  std::cout << "array B (repeating indices)" << std::endl;
  std::cout << "size: " << repeats_count << std::endl;
  if (repeats_count != 0)
    std::cout << "last element:" << find_repeat_gpu_results[repeats_count - 1] << std::endl;

  std::cout << "array C (remaining entries)" << std::endl;
  std::cout << "size: " << remain_count << std::endl;
  if (remain_count != 0)
    std::cout << "last_element: " << remove_repeat_gpu_results[remain_count - 1] << std::endl;

  std::cout << "output exclusive scan gpu results file..." << std::endl;

  std::ofstream myfile1 ("A_exclusive_scan.txt");
  if (myfile1.is_open())
  {
    myfile1 << "size: " << N << "\n";
    for (int i = 0; i < N; i++) {
      myfile1 << exclusive_scan_gpu_results[i] << "\n";
    }
    myfile1.close();
  }

  std::cout << "output repeat indices gpu results file..." << std::endl;

  std::ofstream myfile2 ("B_repeat_indices.txt");
  if (myfile2.is_open())
  {
    myfile2 << "size: " << repeats_count << "\n";
    for (int i = 0; i < repeats_count; i++) {
      myfile2 << find_repeat_gpu_results[i] << "\n";
    }
    myfile2.close();
  }

  std::cout << "output remaining entries gpu results file..." << std::endl;

  std::ofstream myfile3 ("C_remaining_entries.txt");
  if (myfile3.is_open())
  {
    myfile3 << "size: " << remain_count << "\n";
    for (int i = 0; i < remain_count; i++) {
      myfile3 << remove_repeat_gpu_results[i] << "\n";
    }
    myfile3.close();
  }
  
  delete[] randoms;
  delete[] exclusive_scan_check;
  delete[] output_sequential;
  delete[] exclusive_scan_gpu_results;
  delete[] find_repeat_gpu_results;
  delete[] remove_repeat_gpu_results;
  delete[] scan_thrust_results;
  
  return 0;
  
}
